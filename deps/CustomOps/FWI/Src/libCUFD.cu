// Dongzhuo Li 05/06/2018
#include <chrono>
#include <string>
#include "Boundary.h"
#include "Cpml.h"
#include "Model.h"
#include "Parameter.h"
#include "Src_Rec.h"
#include "utilities.h"
using std::string;

/*
        double misfit
        double *grad_Lambda : gradients of Lambda (lame parameter)
        double *grad_Mu : gradients of Mu (shear modulus)
        double *grad_Den : gradients of density
        double *grad_stf : gradients of source time function
        double *Lambda : lame parameter (Mega Pascal)
        double *Mu : shear modulus (Mega Pascal)
        double *Den : density
        double *stf : source time function of all shots
        int calc_id :
                                        calc_id = 0  -- compute residual
                                        calc_id = 1  -- compute gradient
                                        calc_id = 2  -- compute observation only
        int gpu_id  :   CUDA_VISIBLE_DEVICES
        int group_size: number of shots in the group
        int *shot_ids :   processing shot shot_ids
        string para_fname :  parameter path
        // string survey_fname :  survey file (src/rec) path
        // string data_dir : data directory
        // string scratch_dir : temporary files
*/
void cufd(double *misfit, double *grad_Lambda, double *grad_Mu,
          double *grad_Den, double *grad_stf, const double *Lambda,
          const double *Mu, const double *Den, const double *stf, int calc_id,
          const int gpu_id, int group_size, const int *shot_ids,
          const string para_fname) {
  // int deviceCount = 0;
  // CHECK(hipGetDeviceCount (&deviceCount));
  // printf("number of devices = %d\n", deviceCount);
  CHECK(hipSetDevice(gpu_id));
  auto start0 = std::chrono::high_resolution_clock::now();

  if (calc_id < 0 || calc_id > 2) {
    printf("Invalid calc_id %d\n", calc_id);
    exit(0);
  }

  // NOTE Read parameter file
  Parameter para(para_fname, calc_id);
  int nz = para.nz();
  int nx = para.nx();
  int nPml = para.nPoints_pml();
  int nPad = para.nPad();
  float dz = para.dz();
  float dx = para.dx();
  float dt = para.dt();
  float f0 = para.f0();

  int iSnap = 500;  // 400
  int nrec = 1;
  float win_ratio = 0.005;
  int nSteps = para.nSteps();
  float amp_ratio = 1.0;

  // transpose models and convert to float
  float *fLambda, *fMu, *fDen;
  fLambda = (float *)malloc(nz * nx * sizeof(float));
  fMu = (float *)malloc(nz * nx * sizeof(float));
  fDen = (float *)malloc(nz * nx * sizeof(float));
  for (int i = 0; i < nz; i++) {
    for (int j = 0; j < nx; j++) {
      fLambda[j * nz + i] = Lambda[i * nx + j] * MEGA;
      fMu[j * nz + i] = Mu[i * nx + j] * MEGA;
      fDen[j * nz + i] = Den[i * nx + j];
    }
  }
  Model model(para, fLambda, fMu, fDen);
  // Model model;
  Cpml cpml(para, model);
  Bnd boundaries(para);

  auto startSrc = std::chrono::high_resolution_clock::now();
  Src_Rec src_rec(para, para.survey_fname(), stf, group_size, shot_ids);
  // TODO: group_size -> shot group size
  auto finishSrc = std::chrono::high_resolution_clock::now();
#ifdef VERBOSE
  std::chrono::duration<double> elapsedSrc = finishSrc - startSrc;
  std::cout << "Src_Rec time: " << elapsedSrc.count() << " second(s)"
            << std::endl;
  std::cout << "number of shots " << src_rec.d_vec_z_rec.size() << std::endl;
#endif

  // compute Courant number
  compCourantNumber(model.h_Cp, nz * nx, dt, dz, dx);

  dim3 threads(TX, TY);
  dim3 blocks((nz + TX - 1) / TX, (nx + TY - 1) / TY);
  // dim3 threads2(TX + 4, TY + 4);
  // dim3 blocks2((nz + TX + 3) / (TX + 4), (nx + TY + 3) / (TY + 4));

  float *d_vz, *d_vx, *d_szz, *d_sxx, *d_sxz, *d_vz_adj, *d_vx_adj, *d_szz_adj,
      *d_sxx_adj, *d_sxz_adj;
  float *d_mem_dvz_dz, *d_mem_dvz_dx, *d_mem_dvx_dz, *d_mem_dvx_dx;
  float *d_mem_dszz_dz, *d_mem_dsxx_dx, *d_mem_dsxz_dz, *d_mem_dsxz_dx;
  float *d_l2Obj_temp;
  float *h_l2Obj_temp = nullptr;
  h_l2Obj_temp = (float *)malloc(sizeof(float));
  float h_l2Obj = 0.0;
  float *d_gauss_amp;
  float *d_data;
  float *d_data_obs;
  float *d_res;
  CHECK(hipMalloc((void **)&d_vz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_vx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_szz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_sxx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_sxz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_vz_adj, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_vx_adj, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_szz_adj, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_sxx_adj, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_sxz_adj, nz * nx * sizeof(float)));

  CHECK(hipMalloc((void **)&d_mem_dvz_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dvz_dx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dvx_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dvx_dx, nz * nx * sizeof(float)));

  CHECK(hipMalloc((void **)&d_mem_dszz_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dsxx_dx, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dsxz_dz, nz * nx * sizeof(float)));
  CHECK(hipMalloc((void **)&d_mem_dsxz_dx, nz * nx * sizeof(float)));

  CHECK(hipMalloc((void **)&d_l2Obj_temp, 1 * sizeof(float)));
  CHECK(hipMalloc((void **)&d_gauss_amp, 81 * sizeof(float)));
  src_rec_gauss_amp<<<1, threads>>>(d_gauss_amp, 9, 9);

  float *h_snap, *h_snap_back, *h_snap_adj;
  h_snap = (float *)malloc(nz * nx * sizeof(float));
  h_snap_back = (float *)malloc(nz * nx * sizeof(float));
  h_snap_adj = (float *)malloc(nz * nx * sizeof(float));

  hipStream_t streams[group_size];

  auto finish0 = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed0 = finish0 - start0;
#ifdef VERBOSE
  std::cout << "Initialization time: " << elapsed0.count() << " second(s)"
            << std::endl;
#endif

  auto start = std::chrono::high_resolution_clock::now();

  // NOTE Processing Shot
  for (int iShot = 0; iShot < group_size; iShot++) {
#ifdef VERBOSE
    printf("	Processing shot %d\n", shot_ids[iShot]);
#endif
    CHECK(hipStreamCreate(&streams[iShot]));

    intialArrayGPU<<<blocks, threads>>>(d_vz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_vx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_vz_adj, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_vx_adj, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_szz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_sxx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_sxz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_szz_adj, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_sxx_adj, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_sxz_adj, nz, nx, 0.0);

    intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dx, nz, nx, 0.0);

    intialArrayGPU<<<blocks, threads>>>(d_mem_dszz_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dsxx_dx, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dz, nz, nx, 0.0);
    intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dx, nz, nx, 0.0);

    nrec = src_rec.vec_nrec.at(iShot);

    CHECK(hipMalloc((void **)&d_data, nrec * nSteps * sizeof(float)));
    intialArrayGPU<<<blocks, threads>>>(d_data, nrec, nSteps, 0.0);

    if (para.if_res()) {
      fileBinLoad(src_rec.vec_data_obs.at(iShot), nSteps * nrec,
                  para.data_dir_name() + "/Shot" +
                      std::to_string(shot_ids[iShot]) + ".bin");
      CHECK(hipMalloc((void **)&d_data_obs, nrec * nSteps * sizeof(float)));
      CHECK(hipMalloc((void **)&d_res, nrec * nSteps * sizeof(float)));
      intialArrayGPU<<<blocks, threads>>>(d_data_obs, nrec, nSteps, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_res, nrec, nSteps, 0.0);
      CHECK(hipMemcpyAsync(d_data_obs, src_rec.vec_data_obs.at(iShot),
                            nrec * nSteps * sizeof(float),
                            hipMemcpyHostToDevice, streams[iShot]));
    }
    // ------------------------ time loop ----------------------------
    for (int it = 0; it <= nSteps - 2; it++) {
      // ================= elastic =====================
      if (para.withAdj()) {
        // save and record from the beginning
        boundaries.field_from_bnd(d_szz, d_sxz, d_sxx, d_vz, d_vx, it);
      }

      // get snapshot at time it
      if (it == iSnap && iShot == 0) {
        CHECK(hipMemcpy(h_snap, d_vx, nz * nx * sizeof(float),
                         hipMemcpyDeviceToHost));
      }

      el_stress<<<blocks, threads>>>(
          d_vz, d_vx, d_szz, d_sxx, d_sxz, d_mem_dvz_dz, d_mem_dvz_dx,
          d_mem_dvx_dz, d_mem_dvx_dx, model.d_Lambda, model.d_Mu,
          model.d_ave_Mu, model.d_Den, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z,
          cpml.d_K_z_half, cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x,
          cpml.d_a_x, cpml.d_b_x, cpml.d_K_x_half, cpml.d_a_x_half,
          cpml.d_b_x_half, nz, nx, dt, dz, dx, nPml, nPad, true, d_szz_adj,
          d_sxx_adj, d_sxz_adj, model.d_LambdaGrad, model.d_MuGrad);

      add_source<<<1, threads>>>(d_szz, d_sxx, src_rec.vec_source.at(iShot)[it],
                                 nz, true, src_rec.vec_z_src.at(iShot),
                                 src_rec.vec_x_src.at(iShot), dt, d_gauss_amp);

      el_velocity<<<blocks, threads>>>(
          d_vz, d_vx, d_szz, d_sxx, d_sxz, d_mem_dszz_dz, d_mem_dsxz_dx,
          d_mem_dsxz_dz, d_mem_dsxx_dx, model.d_Lambda, model.d_Mu,
          model.d_ave_Byc_a, model.d_ave_Byc_b, cpml.d_K_z, cpml.d_a_z,
          cpml.d_b_z, cpml.d_K_z_half, cpml.d_a_z_half, cpml.d_b_z_half,
          cpml.d_K_x, cpml.d_a_x, cpml.d_b_x, cpml.d_K_x_half, cpml.d_a_x_half,
          cpml.d_b_x_half, nz, nx, dt, dz, dx, nPml, nPad, true, d_vz_adj,
          d_vx_adj, model.d_DenGrad);

      recording<<<(nrec + 31) / 32, 32>>>(
          d_szz, d_sxx, nz, d_data, iShot, it + 1, nSteps, nrec,
          src_rec.d_vec_z_rec.at(iShot), src_rec.d_vec_x_rec.at(iShot));
    }  // end of forward time loop

    if (!para.if_res()) {
      CHECK(hipMemcpyAsync(src_rec.vec_data.at(iShot), d_data,
                            nSteps * nrec * sizeof(float),
                            hipMemcpyDeviceToHost,
                            streams[iShot]));  // test
    }

#ifdef DEBUG
    fileBinWrite(h_snap, nz * nx, "SnapGPU.bin");
#endif

    // compute residuals
    if (para.if_res()) {
      dim3 blocksT((nSteps + TX - 1) / TX, (nrec + TY - 1) / TY);

      // windowing
      if (para.if_win()) {
        cuda_window<<<blocksT, threads>>>(
            nSteps, nrec, dt, src_rec.d_vec_win_start.at(iShot),
            src_rec.d_vec_win_end.at(iShot), src_rec.d_vec_weights.at(iShot),
            win_ratio, d_data_obs);
        cuda_window<<<blocksT, threads>>>(
            nSteps, nrec, dt, src_rec.d_vec_win_start.at(iShot),
            src_rec.d_vec_win_end.at(iShot), src_rec.d_vec_weights.at(iShot),
            win_ratio, d_data);
      } else {
        cuda_window<<<blocksT, threads>>>(nSteps, nrec, dt, win_ratio,
                                          d_data_obs);
        cuda_window<<<blocksT, threads>>>(nSteps, nrec, dt, win_ratio, d_data);
      }

      // filtering
      if (para.if_filter()) {
        bp_filter1d(nSteps, dt, nrec, d_data_obs, para.filter());
        bp_filter1d(nSteps, dt, nrec, d_data, para.filter());
      }

      // Calculate source update and filter calculated data
      if (para.if_src_update()) {
        amp_ratio =
            source_update(nSteps, dt, nrec, d_data_obs, d_data,
                          src_rec.d_vec_source.at(iShot), src_rec.d_coef);
        printf("	Source update => Processing shot %d, amp_ratio = %f\n",
               iShot, amp_ratio);
      }
      amp_ratio = 1.0;  // amplitude not used, so set to 1.0

      // objective function
      gpuMinus<<<blocksT, threads>>>(d_res, d_data_obs, d_data, nSteps, nrec);
      cuda_cal_objective<<<1, 512>>>(d_l2Obj_temp, d_res, nSteps * nrec);
      CHECK(hipMemcpy(h_l2Obj_temp, d_l2Obj_temp, sizeof(float),
                       hipMemcpyDeviceToHost));
      h_l2Obj += h_l2Obj_temp[0];

      //  update source again (adjoint)
      if (para.if_src_update()) {
        source_update_adj(nSteps, dt, nrec, d_res, amp_ratio, src_rec.d_coef);
      }

      // filtering again (adjoint)
      if (para.if_filter()) {
        bp_filter1d(nSteps, dt, nrec, d_res, para.filter());
      }
      // windowing again (adjoint)
      if (para.if_win()) {
        cuda_window<<<blocksT, threads>>>(
            nSteps, nrec, dt, src_rec.d_vec_win_start.at(iShot),
            src_rec.d_vec_win_end.at(iShot), src_rec.d_vec_weights.at(iShot),
            0.1, d_res);
      } else {
        cuda_window<<<blocksT, threads>>>(nSteps, nrec, dt, win_ratio, d_res);
      }

      CHECK(hipMemcpyAsync(src_rec.vec_res.at(iShot), d_res,
                            nSteps * nrec * sizeof(float),
                            hipMemcpyDeviceToHost,
                            streams[iShot]));  // test
      CHECK(hipMemcpyAsync(src_rec.vec_data.at(iShot), d_data,
                            nSteps * nrec * sizeof(float),
                            hipMemcpyDeviceToHost,
                            streams[iShot]));  // test
      CHECK(hipMemcpyAsync(src_rec.vec_data_obs.at(iShot), d_data_obs,
                            nSteps * nrec * sizeof(float),
                            hipMemcpyDeviceToHost,
                            streams[iShot]));  // save preconditioned observed
      CHECK(hipMemcpy(src_rec.vec_source.at(iShot),
                       src_rec.d_vec_source.at(iShot), nSteps * sizeof(float),
                       hipMemcpyDeviceToHost));
    }
    // =================
    hipDeviceSynchronize();

    if (para.withAdj()) {
      // --------------------- Backward ----------------------------
      // initialization
      intialArrayGPU<<<blocks, threads>>>(d_vz_adj, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_vx_adj, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_szz_adj, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_sxx_adj, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_sxz_adj, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dvz_dx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dvx_dx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dszz_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dsxz_dz, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(d_mem_dsxx_dx, nz, nx, 0.0);
      intialArrayGPU<<<blocks, threads>>>(model.d_StfGrad, nSteps, 1, 0.0);
      initialArray(model.h_StfGrad, nSteps, 0.0);

      el_velocity_adj<<<blocks, threads>>>(
          d_vz_adj, d_vx_adj, d_szz_adj, d_sxx_adj, d_sxz_adj, d_mem_dszz_dz,
          d_mem_dsxz_dx, d_mem_dsxz_dz, d_mem_dsxx_dx, d_mem_dvz_dz,
          d_mem_dvz_dx, d_mem_dvx_dz, d_mem_dvx_dx, model.d_Lambda, model.d_Mu,
          model.d_ave_Mu, model.d_Den, model.d_ave_Byc_a, model.d_ave_Byc_b,
          cpml.d_K_z_half, cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x_half,
          cpml.d_a_x_half, cpml.d_b_x_half, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z,
          cpml.d_K_x, cpml.d_a_x, cpml.d_b_x, nz, nx, dt, dz, dx, nPml, nPad);

      res_injection<<<(nrec + 31) / 32, 32>>>(
          d_szz_adj, d_sxx_adj, nz, d_res, nSteps - 1, dt, nSteps, nrec,
          src_rec.d_vec_z_rec.at(iShot), src_rec.d_vec_x_rec.at(iShot));

      el_stress_adj<<<blocks, threads>>>(
          d_vz_adj, d_vx_adj, d_szz_adj, d_sxx_adj, d_sxz_adj, d_mem_dszz_dz,
          d_mem_dsxz_dx, d_mem_dsxz_dz, d_mem_dsxx_dx, d_mem_dvz_dz,
          d_mem_dvz_dx, d_mem_dvx_dz, d_mem_dvx_dx, model.d_Lambda, model.d_Mu,
          model.d_ave_Mu, model.d_Den, model.d_ave_Byc_a, model.d_ave_Byc_b,
          cpml.d_K_z_half, cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x_half,
          cpml.d_a_x_half, cpml.d_b_x_half, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z,
          cpml.d_K_x, cpml.d_a_x, cpml.d_b_x, nz, nx, dt, dz, dx, nPml, nPad);
      for (int it = nSteps - 2; it >= 0; it--) {
        // source time function kernels
        source_grad<<<1, 1>>>(d_szz_adj, d_sxx_adj, nz, model.d_StfGrad, it, dt,
                              src_rec.vec_z_src.at(iShot),
                              src_rec.vec_x_src.at(iShot));

        el_velocity<<<blocks, threads>>>(
            d_vz, d_vx, d_szz, d_sxx, d_sxz, d_mem_dszz_dz, d_mem_dsxz_dx,
            d_mem_dsxz_dz, d_mem_dsxx_dx, model.d_Lambda, model.d_Mu,
            model.d_ave_Byc_a, model.d_ave_Byc_b, cpml.d_K_z, cpml.d_a_z,
            cpml.d_b_z, cpml.d_K_z_half, cpml.d_a_z_half, cpml.d_b_z_half,
            cpml.d_K_x, cpml.d_a_x, cpml.d_b_x, cpml.d_K_x_half,
            cpml.d_a_x_half, cpml.d_b_x_half, nz, nx, dt, dz, dx, nPml, nPad,
            false, d_vz_adj, d_vx_adj, model.d_DenGrad);
        boundaries.field_to_bnd(d_szz, d_sxz, d_sxx, d_vz, d_vx, it, false);

        add_source<<<1, threads>>>(
            d_szz, d_sxx, src_rec.vec_source.at(iShot)[it], nz, false,
            src_rec.vec_z_src.at(iShot), src_rec.vec_x_src.at(iShot), dt,
            d_gauss_amp);

        el_stress<<<blocks, threads>>>(
            d_vz, d_vx, d_szz, d_sxx, d_sxz, d_mem_dvz_dz, d_mem_dvz_dx,
            d_mem_dvx_dz, d_mem_dvx_dx, model.d_Lambda, model.d_Mu,
            model.d_ave_Mu, model.d_Den, cpml.d_K_z, cpml.d_a_z, cpml.d_b_z,
            cpml.d_K_z_half, cpml.d_a_z_half, cpml.d_b_z_half, cpml.d_K_x,
            cpml.d_a_x, cpml.d_b_x, cpml.d_K_x_half, cpml.d_a_x_half,
            cpml.d_b_x_half, nz, nx, dt, dz, dx, nPml, nPad, false, d_szz_adj,
            d_sxx_adj, d_sxz_adj, model.d_LambdaGrad, model.d_MuGrad);
        boundaries.field_to_bnd(d_szz, d_sxz, d_sxx, d_vz, d_vx, it, true);

        el_velocity_adj<<<blocks, threads>>>(
            d_vz_adj, d_vx_adj, d_szz_adj, d_sxx_adj, d_sxz_adj, d_mem_dszz_dz,
            d_mem_dsxz_dx, d_mem_dsxz_dz, d_mem_dsxx_dx, d_mem_dvz_dz,
            d_mem_dvz_dx, d_mem_dvx_dz, d_mem_dvx_dx, model.d_Lambda,
            model.d_Mu, model.d_ave_Mu, model.d_Den, model.d_ave_Byc_a,
            model.d_ave_Byc_b, cpml.d_K_z_half, cpml.d_a_z_half,
            cpml.d_b_z_half, cpml.d_K_x_half, cpml.d_a_x_half, cpml.d_b_x_half,
            cpml.d_K_z, cpml.d_a_z, cpml.d_b_z, cpml.d_K_x, cpml.d_a_x,
            cpml.d_b_x, nz, nx, dt, dz, dx, nPml, nPad);

        res_injection<<<(nrec + 31) / 32, 32>>>(
            d_szz_adj, d_sxx_adj, nz, d_res, it, dt, nSteps, nrec,
            src_rec.d_vec_z_rec.at(iShot), src_rec.d_vec_x_rec.at(iShot));

        el_stress_adj<<<blocks, threads>>>(
            d_vz_adj, d_vx_adj, d_szz_adj, d_sxx_adj, d_sxz_adj, d_mem_dszz_dz,
            d_mem_dsxz_dx, d_mem_dsxz_dz, d_mem_dsxx_dx, d_mem_dvz_dz,
            d_mem_dvz_dx, d_mem_dvx_dz, d_mem_dvx_dx, model.d_Lambda,
            model.d_Mu, model.d_ave_Mu, model.d_Den, model.d_ave_Byc_a,
            model.d_ave_Byc_b, cpml.d_K_z_half, cpml.d_a_z_half,
            cpml.d_b_z_half, cpml.d_K_x_half, cpml.d_a_x_half, cpml.d_b_x_half,
            cpml.d_K_z, cpml.d_a_z, cpml.d_b_z, cpml.d_K_x, cpml.d_a_x,
            cpml.d_b_x, nz, nx, dt, dz, dx, nPml, nPad);

        if (it == iSnap && iShot == 0) {
          CHECK(hipMemcpy(h_snap_back, d_vz, nz * nx * sizeof(float),
                           hipMemcpyDeviceToHost));
          CHECK(hipMemcpy(h_snap_adj, d_szz_adj, nz * nx * sizeof(float),
                           hipMemcpyDeviceToHost));
        }
        if (iShot == 0) {
          // CHECK(hipMemcpy(h_snap_adj, d_szz_adj, nz * nx * sizeof(float),
          //                  hipMemcpyDeviceToHost));
          // fileBinWrite(h_snap_adj, nz * nx,
          //              "SnapGPU_adj_" + std::to_string(it) + ".bin");
          // CHECK(hipMemcpy(h_snap, d_szz, nz * nx * sizeof(float),
          //                  hipMemcpyDeviceToHost));
          // fileBinWrite(h_snap, nz * nx,
          //              "SnapGPU_" + std::to_string(it) + ".bin");
        }
      }  // the end of backward time loop
#ifdef DEBUG
      fileBinWrite(h_snap_back, nz * nx, "SnapGPU_back.bin");
      fileBinWrite(h_snap_adj, nz * nx, "SnapGPU_adj.bin");
#endif

      // transfer source gradient to cpu
      CHECK(hipMemcpy(model.h_StfGrad, model.d_StfGrad, nSteps * sizeof(float),
                       hipMemcpyDeviceToHost));
      for (int it = 0; it < nSteps; it++) {
        grad_stf[iShot * nSteps + it] = model.h_StfGrad[it];
      }
    }  // end bracket of if adj
    CHECK(hipFree(d_data));
    if (para.if_res()) {
      CHECK(hipFree(d_data_obs));
      CHECK(hipFree(d_res));
    }
  }  // the end of shot loop

  auto finish = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> elapsed = finish - start;
#ifdef VERBOSE
  std::cout << "Elapsed time: " << elapsed.count() << " second(s)."
            << std::endl;
#endif

  if (para.withAdj()) {
    // transfer gradients to cpu
    CHECK(hipMemcpy(model.h_LambdaGrad, model.d_LambdaGrad,
                     nz * nx * sizeof(float), hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(model.h_MuGrad, model.d_MuGrad, nz * nx * sizeof(float),
                     hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(model.h_DenGrad, model.d_DenGrad, nz * nx * sizeof(float),
                     hipMemcpyDeviceToHost));
    for (int i = 0; i < nz; i++) {
      for (int j = 0; j < nx; j++) {
        grad_Lambda[i * nx + j] = model.h_LambdaGrad[j * nz + i];
        grad_Mu[i * nx + j] = model.h_MuGrad[j * nz + i];
        grad_Den[i * nx + j] = model.h_DenGrad[j * nz + i];
      }
    }
#ifdef DEBUG
    fileBinWrite(model.h_LambdaGrad, nz * nx, "LambdaGradient.bin");
    fileBinWrite(model.h_MuGrad, nz * nx, "MuGradient.bin");
    fileBinWrite(model.h_DenGrad, nz * nx, "DenGradient.bin");
#endif

    if (para.if_save_scratch()) {
      for (int iShot = 0; iShot < group_size; iShot++) {
        fileBinWrite(src_rec.vec_res.at(iShot),
                     nSteps * src_rec.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/Residual_Shot" +
                         std::to_string(shot_ids[iShot]) + ".bin");
        fileBinWrite(src_rec.vec_data.at(iShot),
                     nSteps * src_rec.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/Syn_Shot" +
                         std::to_string(shot_ids[iShot]) + ".bin");
        fileBinWrite(src_rec.vec_data_obs.at(iShot),
                     nSteps * src_rec.vec_nrec.at(iShot),
                     para.scratch_dir_name() + "/CondObs_Shot" +
                         std::to_string(shot_ids[iShot]) + ".bin");
        fileBinWrite(src_rec.vec_source.at(iShot), nSteps,
                     para.scratch_dir_name() + "/src_updated" +
                         std::to_string(shot_ids[iShot]) + ".bin");
      }
    }
  }

  if (!para.if_res()) {
    for (int iShot = 0; iShot < group_size; iShot++) {
      fileBinWrite(src_rec.vec_data.at(iShot),
                   nSteps * src_rec.vec_nrec.at(iShot),
                   para.data_dir_name() + "/Shot" +
                       std::to_string(shot_ids[iShot]) + ".bin");
    }
  }

  // #ifdef DEBUG
  //   std::cout << "cufd--" << __LINE__ << std::endl;
  // #endif

  // output residual
  if (para.if_res() && !para.withAdj()) {
    h_l2Obj = 0.5 * h_l2Obj;  // DL 02/21/2019 (need to make misfit accurate
                              // here rather than in the script)
    #ifdef VERBOSE
    std::cout << "Total l2 residual = " << std::to_string(h_l2Obj) << std::endl;
    std::cout << "calc_id = " << calc_id << std::endl;
    #endif
    *misfit = h_l2Obj;
  }

  free(h_l2Obj_temp);

  free(h_snap);

  free(h_snap_back);

  free(h_snap_adj);

  free(fLambda);

  free(fMu);

  free(fDen);

  // destroy the streams
  for (int iShot = 0; iShot < group_size; iShot++)
    CHECK(hipStreamDestroy(streams[iShot]));

  hipFree(d_vz);
  hipFree(d_vx);
  hipFree(d_szz);
  hipFree(d_sxx);
  hipFree(d_sxz);
  hipFree(d_vz_adj);
  hipFree(d_vx_adj);
  hipFree(d_szz_adj);
  hipFree(d_sxx_adj);
  hipFree(d_sxz_adj);
  hipFree(d_mem_dvz_dz);
  hipFree(d_mem_dvz_dx);
  hipFree(d_mem_dvx_dz);
  hipFree(d_mem_dvx_dx);
  hipFree(d_mem_dszz_dz);
  hipFree(d_mem_dsxx_dx);
  hipFree(d_mem_dsxz_dz);
  hipFree(d_mem_dsxz_dx);
  hipFree(d_l2Obj_temp);
  hipFree(d_gauss_amp);

#ifdef VERBOSE
  std::cout << "Done!" << std::endl;
#endif
}
